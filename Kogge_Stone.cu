
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<iostream>
#include<math.h>

__global__ void add(int *a,int len)
{
    __shared__ int mem[1954];
    int tid =blockDim.x*blockIdx.x+threadIdx.x;
    int n=1;
    for(;tid<len && tid<16384*n;tid=tid+16384)
    {
    __syncthreads();
    for(int i=0;i<logf(len);i++)
    {
            
            int j=powf(2,i);
            a[tid+j]=a[tid]+a[tid+j];          //sum of elements   for depth of logflen
        
    }
    mem[n]=a[16384*n];    
    a[tid]+=mem[n];
     __syncthreads();
    n++;
    }
}
int main(void)
{
    int len=32000000;
    int *a_d;
    int *a=(int *)malloc(sizeof(int)*len);
    for(int i=0;i<len;i++)
    {
        a[i]=rand()%10;
    }
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc((int **)&a_d,sizeof(int)*len);
    hipMemcpy(a_d,a,sizeof(int)*len,hipMemcpyHostToDevice);
    dim3 threadsPerBlock(32,4);
    dim3 blocksPerGrid(32,4);
    hipEventRecord(start);
    add<<<blocksPerGrid,threadsPerBlock>>>(a_d,len);
    hipMemcpy(a,a_d,sizeof(int)*len,hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipDeviceSynchronize();
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    printf("Elapsed time is : %f millisec\n\n",milliseconds);    
    hipFree(a_d);
    
}
